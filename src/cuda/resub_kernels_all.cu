#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"
#include <cstdio>
#include <algorithm>
#include <vector>
#include <cstdint>

// Need to include Window definition - must match the real struct layout
namespace fresub {
struct Window {
    int target_node;
    std::vector<int> inputs;     // Window inputs (cut leaves)
    std::vector<int> nodes;      // All nodes in window
    std::vector<int> divisors;   // Window nodes - MFFC(target)
    int cut_id;                  // ID of the cut that generated this window
    int mffc_size;
    std::vector<std::vector<uint64_t>> truth_tables;
    std::vector<std::vector<int>> feasible_combinations;
};
}

#define word_width 64
#define THREADS_PER_PROBLEM 32
#define BLOCK_SIZE 256

// Check for CUDA error macro
#ifndef CHECK_CUDA_ERROR
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(1); \
        } \
    } while(0)
#endif

namespace fresub {
namespace cuda {

// External declaration - function defined in resub_kernels.cu
extern __device__ int solve_resub_overlap_cuda(int i, int j, int k, int l, uint64_t *flat_problem, int nWords, int problem_offset, int n_divs);

// CUDA kernel to mark all feasible 4-combinations as boolean array
// Each thread handles multiple i values for one problem using the same loop structure
__global__ void solve_resub_problems_kernel_all(uint64_t *flat_problems, char *feasibility_results,
                                               int *problem_offsets, int *combination_offsets, 
                                               int *num_inputs, int M) {
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int problem_id = global_tid / THREADS_PER_PROBLEM;
    int thread_in_problem = global_tid % THREADS_PER_PROBLEM;
    
    if (problem_id >= M) return;
    
    int N = num_inputs[problem_id];
    int size = 1 << N;
    int nWords = size / word_width + (size % word_width != 0 ? 1 : 0);
    int problem_offset = problem_offsets[problem_id];
    int n_divs = (problem_offsets[problem_id + 1] - problem_offset) / nWords - 1;
    
    int combination_base = combination_offsets[problem_id];
    
    // Each thread handles multiple values of i using stride pattern (same as original)
    for (int i = thread_in_problem; i < n_divs; i += THREADS_PER_PROBLEM) {
        for (int j = i + 1; j < n_divs; j++) {
            for (int k = j + 1; k < n_divs; k++) {
                for (int l = k + 1; l < n_divs; l++) {
                    // Calculate index in feasibility array using 4D indexing
                    int combination_idx = l + k * n_divs + j * n_divs * n_divs + i * n_divs * n_divs * n_divs;
                    int global_idx = combination_base + combination_idx;
                    
                    uint32_t mask = solve_resub_overlap_cuda(i, j, k, l, flat_problems, nWords, problem_offset, n_divs);
                    feasibility_results[global_idx] = (mask != 0) ? 1 : 0;
                }
            }
        }
    }
}

// Host function to launch new CUDA kernel that finds all feasible combinations
void solve_resub_problems_cuda_all(uint64_t *flat_problems, char *feasibility_results,
                                   int *problem_offsets, int *combination_offsets, 
                                   int *num_inputs, int M, int total_elements, int total_combinations) {
    // Allocate device memory
    uint64_t *d_flat_problems;
    char *d_feasibility_results;
    int *d_problem_offsets;
    int *d_combination_offsets;
    int *d_num_inputs;
    
    CHECK_CUDA_ERROR(hipMalloc(&d_flat_problems, total_elements * sizeof(uint64_t)));
    CHECK_CUDA_ERROR(hipMalloc(&d_feasibility_results, total_combinations * sizeof(char)));
    CHECK_CUDA_ERROR(hipMalloc(&d_problem_offsets, (M + 1) * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_combination_offsets, (M + 1) * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_num_inputs, M * sizeof(int)));
    
    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpy(d_flat_problems, flat_problems, total_elements * sizeof(uint64_t), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_problem_offsets, problem_offsets, (M + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_combination_offsets, combination_offsets, (M + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_num_inputs, num_inputs, M * sizeof(int), hipMemcpyHostToDevice));
    
    // Launch kernel
    int blockSize = BLOCK_SIZE;
    int totalThreads = M * THREADS_PER_PROBLEM;
    int numBlocks = (totalThreads + blockSize - 1) / blockSize;
    
    solve_resub_problems_kernel_all<<<numBlocks, blockSize>>>(d_flat_problems, d_feasibility_results,
                                                              d_problem_offsets, d_combination_offsets, 
                                                              d_num_inputs, M);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    // Copy results back
    CHECK_CUDA_ERROR(hipMemcpy(feasibility_results, d_feasibility_results, total_combinations * sizeof(char), hipMemcpyDeviceToHost));
    
    // Cleanup
    CHECK_CUDA_ERROR(hipFree(d_flat_problems));
    CHECK_CUDA_ERROR(hipFree(d_feasibility_results));
    CHECK_CUDA_ERROR(hipFree(d_problem_offsets));
    CHECK_CUDA_ERROR(hipFree(d_combination_offsets));
    CHECK_CUDA_ERROR(hipFree(d_num_inputs));
}

} // namespace cuda
} // namespace fresub

namespace fresub {

// CUDA-compatible feasibility check function that finds ALL feasible combinations
void feasibility_check_cuda_all(std::vector<Window>::iterator begin, std::vector<Window>::iterator end) {
    int M = std::distance(begin, end);  // Number of problems (windows)
    if (M == 0) return;
    
    // Calculate total size needed for truth tables and build offset arrays
    std::vector<int> problem_offsets(M + 1);  // M+1 elements
    std::vector<int> combination_offsets(M + 1);  // M+1 elements
    std::vector<int> num_inputs(M);
    int total_elements = 0;
    int total_combinations = 0;
    
    int idx = 0;
    for (auto it = begin; it != end; ++it, ++idx) {
        int N = it->inputs.size();
        num_inputs[idx] = N;
        int size = 1 << N;
        int nWords = (size + 63) / 64;  // More standard rounding up
        int n_truth_tables = it->truth_tables.size();  // divisors + target
        int n_divs = n_truth_tables - 1;  // -1 for target
        
        problem_offsets[idx] = total_elements;
        total_elements += n_truth_tables * nWords;
        
        combination_offsets[idx] = total_combinations;
        total_combinations += n_divs * n_divs * n_divs * n_divs;  // D^4 combinations
    }
    problem_offsets[M] = total_elements;  // Last element points to end
    combination_offsets[M] = total_combinations;  // Last element points to end
    
    // Allocate flattened arrays
    std::vector<uint64_t> flat_problems(total_elements, 0);
    std::vector<char> feasibility_results(total_combinations, 0);  // Use char instead of bool
    
    // Flatten all windows into the truth table array
    idx = 0;
    for (auto it = begin; it != end; ++it, ++idx) {
        int N = it->inputs.size();
        int size = 1 << N;
        int nWords = size / 64 + (size % 64 != 0 ? 1 : 0);
        int problem_offset = problem_offsets[idx];
        
        // Copy all truth tables (divisors + target)
        for (size_t t = 0; t < it->truth_tables.size(); t++) {
            for (int w = 0; w < nWords; w++) {
                if (w < (int)it->truth_tables[t].size()) {
                    flat_problems[problem_offset + t * nWords + w] = it->truth_tables[t][w];
                }
            }
        }
    }
    
    // Call CUDA kernel to find all feasible combinations
    cuda::solve_resub_problems_cuda_all(flat_problems.data(), feasibility_results.data(),
                                        problem_offsets.data(), combination_offsets.data(),
                                        num_inputs.data(), M, total_elements, total_combinations);
    
    // Convert results back to feasible combinations for each window
    idx = 0;
    for (auto it = begin; it != end; ++it, ++idx) {
        int n_divs = it->truth_tables.size() - 1;  // -1 for target
        int combination_base = combination_offsets[idx];
        
        it->feasible_combinations.clear();
        
        // Check all valid combinations (i < j < k < l)
        for (int i = 0; i < n_divs; i++) {
            for (int j = i + 1; j < n_divs; j++) {
                for (int k = j + 1; k < n_divs; k++) {
                    for (int l = k + 1; l < n_divs; l++) {
                        int combination_idx = l + k * n_divs + j * n_divs * n_divs + i * n_divs * n_divs * n_divs;
                        int global_idx = combination_base + combination_idx;
                        
                        if (feasibility_results[global_idx]) {
                            it->feasible_combinations.push_back({i, j, k, l});
                        }
                    }
                }
            }
        }
    }
}

} // namespace fresub