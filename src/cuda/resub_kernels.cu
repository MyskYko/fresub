#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"
#include <cstdio>
#include <algorithm>
#include <vector>
#include <cstdint>

// Need to include Window definition - must match the real struct layout
// This is a bit hacky but avoids pulling in the full AIG dependencies
namespace fresub {
struct aigman; // forward declaration for pointer use
struct FeasibleSet {
    std::vector<int> divisor_indices;
    std::vector<int> divisor_nodes;
    std::vector<aigman*> synths;
};
struct Window {
    int target_node;
    std::vector<int> inputs;     // Window inputs (cut leaves)
    std::vector<int> nodes;      // All nodes in window
    std::vector<int> divisors;   // Window nodes - MFFC(target)
    int cut_id;                  // ID of the cut that generated this window
    int mffc_size;
    std::vector<std::vector<uint64_t>> truth_tables;
    std::vector<FeasibleSet> feasible_sets;
};
}

#define word_width 64
#define THREADS_PER_PROBLEM 32
#define BLOCK_SIZE 256

// Check for CUDA error macro
#ifndef CHECK_CUDA_ERROR
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(1); \
        } \
    } while(0)
#endif

namespace fresub {
namespace cuda {

// CUDA kernel version of solve_resub_overlap using flattened array with variable divisor count
__device__ int solve_resub_overlap_cuda(int i, int j, int k, int l, uint64_t *flat_problem, int nWords, int problem_offset, int n_divs) 
{
    uint32_t res = ((1U << i) | (1U << j) | (1U << k) | (1U << l)); 
    uint64_t qs[32] = {0};
    
    for (int h = 0; h < nWords; h++) {
        // Access flattened array: flat_problem[problem_offset + row_id * nWords + word_id]
        uint64_t t_i = flat_problem[problem_offset + i * nWords + h];
        uint64_t t_j = flat_problem[problem_offset + j * nWords + h];
        uint64_t t_k = flat_problem[problem_offset + k * nWords + h];
        uint64_t t_l = flat_problem[problem_offset + l * nWords + h];
        // Target truth table is at position n_divs (last element)
        uint64_t t_on = flat_problem[problem_offset + n_divs * nWords + h];
        uint64_t t_off = ~t_on;  // Compute off-set from on-set like CPU version

        qs[0]  |=  (t_off &  t_i &  t_j) & ( t_k &  t_l);
        qs[1]  |=  (t_on  &  t_i &  t_j) & ( t_k &  t_l);
        qs[2]  |=  (t_off & ~t_i &  t_j) & ( t_k &  t_l);
        qs[3]  |=  (t_on  & ~t_i &  t_j) & ( t_k &  t_l);
        qs[4]  |=  (t_off &  t_i & ~t_j) & ( t_k &  t_l);
        qs[5]  |=  (t_on  &  t_i & ~t_j) & ( t_k &  t_l);
        qs[6]  |=  (t_off & ~t_i & ~t_j) & ( t_k &  t_l);
        qs[7]  |=  (t_on  & ~t_i & ~t_j) & ( t_k &  t_l);
        qs[8]  |=  (t_off &  t_i &  t_j) & (~t_k &  t_l);
        qs[9]  |=  (t_on  &  t_i &  t_j) & (~t_k &  t_l);
        qs[10] |=  (t_off & ~t_i &  t_j) & (~t_k &  t_l);
        qs[11] |=  (t_on  & ~t_i &  t_j) & (~t_k &  t_l);
        qs[12] |=  (t_off &  t_i & ~t_j) & (~t_k &  t_l);
        qs[13] |=  (t_on  &  t_i & ~t_j) & (~t_k &  t_l);
        qs[14] |=  (t_off & ~t_i & ~t_j) & (~t_k &  t_l);
        qs[15] |=  (t_on  & ~t_i & ~t_j) & (~t_k &  t_l);
        qs[16] |=  (t_off &  t_i &  t_j) & ( t_k & ~t_l);
        qs[17] |=  (t_on  &  t_i &  t_j) & ( t_k & ~t_l);
        qs[18] |=  (t_off & ~t_i &  t_j) & ( t_k & ~t_l);
        qs[19] |=  (t_on  & ~t_i &  t_j) & ( t_k & ~t_l);
        qs[20] |=  (t_off &  t_i & ~t_j) & ( t_k & ~t_l);
        qs[21] |=  (t_on  &  t_i & ~t_j) & ( t_k & ~t_l);
        qs[22] |=  (t_off & ~t_i & ~t_j) & ( t_k & ~t_l);
        qs[23] |=  (t_on  & ~t_i & ~t_j) & ( t_k & ~t_l);
        qs[24] |=  (t_off &  t_i &  t_j) & (~t_k & ~t_l);
        qs[25] |=  (t_on  &  t_i &  t_j) & (~t_k & ~t_l);
        qs[26] |=  (t_off & ~t_i &  t_j) & (~t_k & ~t_l);
        qs[27] |=  (t_on  & ~t_i &  t_j) & (~t_k & ~t_l);
        qs[28] |=  (t_off &  t_i & ~t_j) & (~t_k & ~t_l);
        qs[29] |=  (t_on  &  t_i & ~t_j) & (~t_k & ~t_l);
        qs[30] |=  (t_off & ~t_i & ~t_j) & (~t_k & ~t_l);
        qs[31] |=  (t_on  & ~t_i & ~t_j) & (~t_k & ~t_l);
    }
    
    for (int h = 0; h < 16; h++) {
        int fail = ((qs[2*h] != 0) && (qs[2*h+1] != 0));
        res = fail ? 0 : res; // resub fails
    }
    return res;
}

// CUDA kernel: Thread I handles i=I%THREADS_PER_PROBLEM for problem I/THREADS_PER_PROBLEM
// Now supports variable number of divisors and inputs per problem with no wasted space
// problem_offsets has M+1 elements where problem_offsets[M] = total_elements
__global__ void solve_resub_problems_kernel(uint64_t *flat_problems, uint32_t *solutions, 
                                           int *problem_offsets, int *num_inputs, int M) {
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int problem_id = global_tid / THREADS_PER_PROBLEM;
    int thread_in_problem = global_tid % THREADS_PER_PROBLEM;
    
    if (problem_id >= M) return;
    
    int N = num_inputs[problem_id];  // Get number of inputs for this problem
    int size = 1 << N;
    int nWords = size / word_width + (size % word_width != 0 ? 1 : 0);
    int problem_offset = problem_offsets[problem_id];  // Get offset where this problem's data starts
    
    // Compute divisor count from offsets (problem_offsets[M] contains total_elements)
    int n_divs = (problem_offsets[problem_id + 1] - problem_offset) / nWords - 1;  // -1 for target
    
    uint32_t local_res = 0;
    
    // Each thread handles multiple values of i using stride pattern
    // Loop only up to actual number of divisors for this problem
    for (int i = thread_in_problem; i < n_divs; i += THREADS_PER_PROBLEM) {
        for (int j = i + 1; j < n_divs; j++) {
            for (int k = j + 1; k < n_divs; k++) {
                for (int l = k + 1; l < n_divs; l++) {
                    uint32_t temp = solve_resub_overlap_cuda(i, j, k, l, flat_problems, nWords, problem_offset, n_divs);
                    local_res = local_res ? local_res : temp;
                }
            }
        }
    }
    
    // Reduction within threads working on same problem
    __shared__ uint32_t shared_results[BLOCK_SIZE];
    int shared_idx = threadIdx.x;
    shared_results[shared_idx] = local_res;
    __syncthreads();
    
    // First thread of each problem group performs reduction
    if (thread_in_problem == 0) {
        uint32_t final_res = 0;
        for (int t = 0; t < THREADS_PER_PROBLEM; t++) {
            int idx = (threadIdx.x / THREADS_PER_PROBLEM) * THREADS_PER_PROBLEM + t;
            final_res = final_res ? final_res : shared_results[idx];
        }
        solutions[problem_id] = final_res;
    }
}


// Host function to launch CUDA kernel - takes pre-flattened array with offsets
// problem_offsets has M+1 elements where problem_offsets[M] = total_elements
void solve_resub_problems_cuda(uint64_t *flat_problems, uint32_t *solutions, 
                              int *problem_offsets, int *num_inputs, int M, int total_elements) {
    // Allocate device memory
    uint64_t *d_flat_problems;
    uint32_t *d_solutions;
    int *d_problem_offsets;
    int *d_num_inputs;
    
    CHECK_CUDA_ERROR(hipMalloc(&d_flat_problems, total_elements * sizeof(uint64_t)));
    CHECK_CUDA_ERROR(hipMalloc(&d_solutions, M * sizeof(uint32_t)));
    CHECK_CUDA_ERROR(hipMalloc(&d_problem_offsets, (M + 1) * sizeof(int)));  // M+1 elements
    CHECK_CUDA_ERROR(hipMalloc(&d_num_inputs, M * sizeof(int)));
    
    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpy(d_flat_problems, flat_problems, total_elements * sizeof(uint64_t), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_problem_offsets, problem_offsets, (M + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_num_inputs, num_inputs, M * sizeof(int), hipMemcpyHostToDevice));
    
    // Launch kernel - adjust for multiple threads per problem
    int blockSize = BLOCK_SIZE;
    int totalThreads = M * THREADS_PER_PROBLEM;
    int numBlocks = (totalThreads + blockSize - 1) / blockSize;
    
    solve_resub_problems_kernel<<<numBlocks, blockSize>>>(d_flat_problems, d_solutions, 
                                                          d_problem_offsets, d_num_inputs, M);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    // Copy results back
    CHECK_CUDA_ERROR(hipMemcpy(solutions, d_solutions, M * sizeof(uint32_t), hipMemcpyDeviceToHost));
    
    // Cleanup
    CHECK_CUDA_ERROR(hipFree(d_flat_problems));
    CHECK_CUDA_ERROR(hipFree(d_solutions));
    CHECK_CUDA_ERROR(hipFree(d_problem_offsets));
    CHECK_CUDA_ERROR(hipFree(d_num_inputs));
}

// Convert solution mask to vector of divisor indices
std::vector<int> mask_to_indices(uint32_t mask) {
    std::vector<int> indices;
    for (int i = 0; i < 32; i++) {
        if (mask & (1U << i)) {
            indices.push_back(i);
        }
    }
    return indices;
}

} // namespace cuda
} // namespace fresub

namespace fresub {

// CUDA-compatible feasibility check function with vector iterator interface (original version)
void feasibility_check_cuda(std::vector<Window>::iterator begin, std::vector<Window>::iterator end) {
    int M = std::distance(begin, end);  // Number of problems (windows)
    if (M == 0) return;
    
    // Calculate total size needed and build offset array
    std::vector<int> problem_offsets(M + 1);  // M+1 elements
    std::vector<int> num_inputs(M);
    int total_elements = 0;
    
    int idx = 0;
    for (auto it = begin; it != end; ++it, ++idx) {
        int N = it->inputs.size();
        num_inputs[idx] = N;
        int size = 1 << N;
        int nWords = (size + 63) / 64;  // More standard rounding up
        int n_truth_tables = it->truth_tables.size();  // divisors + target
        
        problem_offsets[idx] = total_elements;
        total_elements += n_truth_tables * nWords;
    }
    problem_offsets[M] = total_elements;  // Last element points to end
    
    // Allocate flattened array with exact size needed
    std::vector<uint64_t> flat_problems(total_elements, 0);
    
    // Flatten all windows into the array
    idx = 0;
    for (auto it = begin; it != end; ++it, ++idx) {
        int N = it->inputs.size();
        int size = 1 << N;
        int nWords = size / 64 + (size % 64 != 0 ? 1 : 0);
        int problem_offset = problem_offsets[idx];
        
        // Copy all truth tables (divisors + target)
        for (size_t t = 0; t < it->truth_tables.size(); t++) {
            for (int w = 0; w < nWords; w++) {
                if (w < (int)it->truth_tables[t].size()) {
                    flat_problems[problem_offset + t * nWords + w] = it->truth_tables[t][w];
                }
            }
        }
    }
    
    // Allocate solutions array
    std::vector<uint32_t> solutions(M);
    
    // Call CUDA kernel with original interface (finds first feasible solution only)
    cuda::solve_resub_problems_cuda(flat_problems.data(), solutions.data(), 
                                    problem_offsets.data(), num_inputs.data(), 
                                    M, total_elements);
    
    // Convert results back to feasible sets for each window
    idx = 0;
    for (auto it = begin; it != end; ++it, ++idx) {
        uint32_t mask = solutions[idx];
        
        if (mask != 0) {
            // Convert mask to indices and store in window - assume indices are valid
            std::vector<int> indices = cuda::mask_to_indices(mask);
            // Populate feasible_sets with divisor indices only
            FeasibleSet fs;
            fs.divisor_indices = std::move(indices);
            it->feasible_sets.push_back(std::move(fs));
        }
    }
}

} // namespace fresub
