#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"
#include <cstdio>
#include <algorithm>

namespace fresub {
namespace cuda {

// Device function to check if a function can be implemented with given divisors
__device__ bool check_implication(
    const uint64_t* on_set,
    const uint64_t* off_set,
    const uint64_t* impl_func,
    int num_words
) {
    // Check that impl_func covers on_set and doesn't intersect off_set
    for (int i = 0; i < num_words; i++) {
        // Must cover all on-set minterms
        if ((on_set[i] & ~impl_func[i]) != 0) {
            return false;
        }
        // Must not cover any off-set minterms
        if ((off_set[i] & impl_func[i]) != 0) {
            return false;
        }
    }
    return true;
}

// Find best subset of divisors that implements the target function
__device__ uint32_t find_best_divisors(
    const uint64_t* divisor_truths,
    const uint64_t* target_on,
    const uint64_t* target_off,
    int num_divisors,
    int num_words,
    int max_size
) {
    uint32_t best_mask = 0;
    
    // Try all combinations up to max_size divisors
    // This is simplified - in practice we'd use more sophisticated search
    for (int size = 1; size <= max_size && size <= 4; size++) {
        // Generate all k-combinations
        if (size == 1) {
            for (int i = 0; i < num_divisors; i++) {
                if (check_implication(target_on, target_off, 
                                     &divisor_truths[i * num_words], num_words)) {
                    return (1U << i);
                }
            }
        } else if (size == 2) {
            for (int i = 0; i < num_divisors; i++) {
                for (int j = i + 1; j < num_divisors; j++) {
                    // Compute AND of two divisors
                    uint64_t combined[MAX_TRUTH_WORDS];
                    for (int w = 0; w < num_words; w++) {
                        combined[w] = divisor_truths[i * num_words + w] & 
                                     divisor_truths[j * num_words + w];
                    }
                    if (check_implication(target_on, target_off, combined, num_words)) {
                        return (1U << i) | (1U << j);
                    }
                    
                    // Try OR as well
                    for (int w = 0; w < num_words; w++) {
                        combined[w] = divisor_truths[i * num_words + w] | 
                                     divisor_truths[j * num_words + w];
                    }
                    if (check_implication(target_on, target_off, combined, num_words)) {
                        return (1U << i) | (1U << j);
                    }
                }
            }
        } else if (size == 3) {
            for (int i = 0; i < num_divisors - 2; i++) {
                for (int j = i + 1; j < num_divisors - 1; j++) {
                    for (int k = j + 1; k < num_divisors; k++) {
                        // Try various 3-input functions
                        uint64_t combined[MAX_TRUTH_WORDS];
                        
                        // Example: (i & j) | k
                        for (int w = 0; w < num_words; w++) {
                            combined[w] = (divisor_truths[i * num_words + w] & 
                                          divisor_truths[j * num_words + w]) |
                                         divisor_truths[k * num_words + w];
                        }
                        if (check_implication(target_on, target_off, combined, num_words)) {
                            return (1U << i) | (1U << j) | (1U << k);
                        }
                        
                        // Example: i & j & k
                        for (int w = 0; w < num_words; w++) {
                            combined[w] = divisor_truths[i * num_words + w] & 
                                         divisor_truths[j * num_words + w] &
                                         divisor_truths[k * num_words + w];
                        }
                        if (check_implication(target_on, target_off, combined, num_words)) {
                            return (1U << i) | (1U << j) | (1U << k);
                        }
                    }
                }
            }
        } else if (size == 4) {
            // Similar for 4 divisors, but more complex
            // Simplified version here
            for (int i = 0; i < num_divisors - 3; i++) {
                for (int j = i + 1; j < num_divisors - 2; j++) {
                    for (int k = j + 1; k < num_divisors - 1; k++) {
                        for (int l = k + 1; l < num_divisors; l++) {
                            uint64_t combined[MAX_TRUTH_WORDS];
                            
                            // Try: ((i & j) | (k & l))
                            for (int w = 0; w < num_words; w++) {
                                combined[w] = (divisor_truths[i * num_words + w] & 
                                              divisor_truths[j * num_words + w]) |
                                             (divisor_truths[k * num_words + w] &
                                              divisor_truths[l * num_words + w]);
                            }
                            if (check_implication(target_on, target_off, combined, num_words)) {
                                return (1U << i) | (1U << j) | (1U << k) | (1U << l);
                            }
                        }
                    }
                }
            }
        }
    }
    
    return best_mask;
}

// Main kernel for resubstitution feasibility checking
__global__ void resub_feasibility_kernel(
    const GPUResubProblem* problems,
    GPUResubResult* results,
    int num_problems
) {
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int problem_id = global_tid / THREADS_PER_PROBLEM;
    int thread_in_problem = global_tid % THREADS_PER_PROBLEM;
    
    if (problem_id >= num_problems) return;
    
    const GPUResubProblem& problem = problems[problem_id];
    
    // Shared memory for reduction
    __shared__ uint32_t shared_masks[BLOCK_SIZE];
    __shared__ int32_t shared_gains[BLOCK_SIZE];
    
    uint32_t local_mask = 0;
    int32_t local_gain = -1;
    
    // Each thread checks different divisor combinations
    int start_idx = thread_in_problem;
    int stride = THREADS_PER_PROBLEM;
    
    // Simple strategy: each thread checks different starting points
    for (int i = start_idx; i < problem.num_divisors && i < MAX_DIVISORS; i += stride) {
        // Try single divisor first
        if (check_implication(problem.target_on, problem.target_off,
                             &problem.divisor_truths[i * problem.num_words],
                             problem.num_words)) {
            local_mask = (1U << i);
            local_gain = 1;  // Simplified gain calculation
            break;
        }
        
        // Try pairs starting from i
        for (int j = i + 1; j < problem.num_divisors && j < MAX_DIVISORS; j++) {
            uint64_t combined[MAX_TRUTH_WORDS];
            
            // AND combination
            for (int w = 0; w < problem.num_words; w++) {
                combined[w] = problem.divisor_truths[i * problem.num_words + w] &
                             problem.divisor_truths[j * problem.num_words + w];
            }
            
            if (check_implication(problem.target_on, problem.target_off,
                                combined, problem.num_words)) {
                local_mask = (1U << i) | (1U << j);
                local_gain = 1;
                break;
            }
            
            // OR combination
            for (int w = 0; w < problem.num_words; w++) {
                combined[w] = problem.divisor_truths[i * problem.num_words + w] |
                             problem.divisor_truths[j * problem.num_words + w];
            }
            
            if (check_implication(problem.target_on, problem.target_off,
                                combined, problem.num_words)) {
                local_mask = (1U << i) | (1U << j);
                local_gain = 1;
                break;
            }
        }
        
        if (local_mask != 0) break;
    }
    
    // Store in shared memory
    int shared_idx = threadIdx.x;
    shared_masks[shared_idx] = local_mask;
    shared_gains[shared_idx] = local_gain;
    __syncthreads();
    
    // Reduction within threads working on same problem
    if (thread_in_problem == 0) {
        uint32_t final_mask = 0;
        int32_t final_gain = -1;
        
        for (int t = 0; t < THREADS_PER_PROBLEM && 
             (threadIdx.x / THREADS_PER_PROBLEM) * THREADS_PER_PROBLEM + t < blockDim.x; t++) {
            int idx = (threadIdx.x / THREADS_PER_PROBLEM) * THREADS_PER_PROBLEM + t;
            if (shared_masks[idx] != 0 && 
                (final_mask == 0 || __popc(shared_masks[idx]) < __popc(final_mask))) {
                final_mask = shared_masks[idx];
                final_gain = shared_gains[idx];
            }
        }
        
        results[problem_id].window_id = problem.window_id;
        results[problem_id].success = (final_mask != 0) ? 1 : 0;
        results[problem_id].divisor_mask = final_mask;
        results[problem_id].gain = final_gain;
    }
}

// Extended kernel for more complex resubstitution
__global__ void parallel_resub_kernel(
    const GPUResubProblem* problems,
    GPUResubResult* results,
    int num_problems,
    int max_divisor_size
) {
    int problem_id = blockIdx.x;
    if (problem_id >= num_problems) return;
    
    const GPUResubProblem& problem = problems[problem_id];
    int tid = threadIdx.x;
    
    // Use shared memory for divisor truth tables
    extern __shared__ uint64_t shared_truths[];
    
    // Cooperatively load divisor truth tables into shared memory
    int truths_per_thread = (problem.num_divisors * problem.num_words + blockDim.x - 1) / blockDim.x;
    for (int i = 0; i < truths_per_thread; i++) {
        int idx = tid * truths_per_thread + i;
        if (idx < problem.num_divisors * problem.num_words) {
            shared_truths[idx] = problem.divisor_truths[idx];
        }
    }
    __syncthreads();
    
    // Each thread explores different combinations
    uint32_t best_mask = find_best_divisors(
        shared_truths,
        problem.target_on,
        problem.target_off,
        problem.num_divisors,
        problem.num_words,
        max_divisor_size
    );
    
    // Reduction to find best result
    __shared__ uint32_t best_masks[256];
    best_masks[tid] = best_mask;
    __syncthreads();
    
    // Tree reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            uint32_t mask1 = best_masks[tid];
            uint32_t mask2 = best_masks[tid + stride];
            
            // Choose better mask (fewer divisors)
            if (mask2 != 0 && (mask1 == 0 || __popc(mask2) < __popc(mask1))) {
                best_masks[tid] = mask2;
            }
        }
        __syncthreads();
    }
    
    // Thread 0 writes the result
    if (tid == 0) {
        results[problem_id].window_id = problem.window_id;
        results[problem_id].success = (best_masks[0] != 0) ? 1 : 0;
        results[problem_id].divisor_mask = best_masks[0];
        results[problem_id].gain = best_masks[0] ? __popc(best_masks[0]) : -1;
    }
}

// GPUResubEngine implementation
GPUResubEngine::GPUResubEngine(int max_batch_size) 
    : max_batch_size(max_batch_size), d_problems(nullptr), d_results(nullptr) {
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate_device_memory(max_batch_size);
}

GPUResubEngine::~GPUResubEngine() {
    free_device_memory();
    hipStreamDestroy(stream);
}

void GPUResubEngine::allocate_device_memory(int batch_size) {
    if (d_problems != nullptr) {
        free_device_memory();
    }
    
    CUDA_CHECK(hipMalloc(&d_problems, batch_size * sizeof(GPUResubProblem)));
    CUDA_CHECK(hipMalloc(&d_results, batch_size * sizeof(GPUResubResult)));
}

void GPUResubEngine::free_device_memory() {
    if (d_problems != nullptr) {
        hipFree(d_problems);
        d_problems = nullptr;
    }
    if (d_results != nullptr) {
        hipFree(d_results);
        d_results = nullptr;
    }
}

void GPUResubEngine::process_batch(
    const GPUResubProblem* h_problems,
    GPUResubResult* h_results,
    int num_problems
) {
    if (num_problems > max_batch_size) {
        // Process in chunks
        int offset = 0;
        while (offset < num_problems) {
            int chunk_size = std::min(max_batch_size, num_problems - offset);
            process_batch(h_problems + offset, h_results + offset, chunk_size);
            offset += chunk_size;
        }
        return;
    }
    
    // Copy problems to device
    CUDA_CHECK(hipMemcpyAsync(d_problems, h_problems, 
                               num_problems * sizeof(GPUResubProblem),
                               hipMemcpyHostToDevice, stream));
    
    // Launch kernel
    int total_threads = num_problems * THREADS_PER_PROBLEM;
    int num_blocks = (total_threads + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    resub_feasibility_kernel<<<num_blocks, BLOCK_SIZE, 0, stream>>>(
        d_problems, d_results, num_problems
    );
    
    // Copy results back
    CUDA_CHECK(hipMemcpyAsync(h_results, d_results,
                               num_problems * sizeof(GPUResubResult),
                               hipMemcpyDeviceToHost, stream));
    
    // Synchronize
    CUDA_CHECK(hipStreamSynchronize(stream));
}

} // namespace cuda
} // namespace fresub